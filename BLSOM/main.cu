#include<iostream>
#include"BLSOM.h"
#include"SelectGPU.h"
#include"LoadDataSet.h"
#include<hiprand/hiprand_kernel.h>
#include<algorithm>

#define MAP_WIDTH 200
#define MAP_HEIGHT 50
#define TRAIN_NUM 200
#define EPOC_NUM 0

int WriteSOMMAP(std::string fileName, float* map, int map_vec, int map_width, int map_height) {
	std::ofstream ofs;
	ofs.open(fileName, 'w');

	if (!ofs) {
		std::cerr << "can't opne file" << std::endl;
		return EXIT_FAILURE;
	}

	ofs << map_vec << std::endl;
	ofs << map_width << std::endl;
	ofs << map_height << std::endl;

	for (int i = 1; i < map_height*map_width; i++) {
		for (int v = 0; v < map_vec; v++) {
			ofs << *map << " ";
			map++;
		}
		ofs << "\n";
	}
	ofs.close();

	return EXIT_SUCCESS;
}

int WriteUmatrix(std::string fileName, std::vector<std::vector<float>> umatrix) {
	std::ofstream ofs;
	ofs.open(fileName, 'w');

	if (!ofs) {
		std::cerr << "can't opne file" << std::endl;
		return EXIT_FAILURE;
	}

	for (int h = 0; h < umatrix.size()-1; h++) {
		for (int w = 0; w < umatrix[0].size()-1; w++) {
			ofs << umatrix[h][w];
			if (w != umatrix[0].size() - 2)
				ofs << "\t";
		}
		if (h != umatrix.size() - 2)
			ofs << "\n";
	}
	ofs.close();

	return EXIT_SUCCESS;
}

int main(int argc, char** argv) {
	int device;
	int vec_dim;
	int map_width;
	int map_height;
	float* som;
	std::vector<std::vector<float>> umatrix;

	std::shared_ptr<float> map_weight;
	std::vector<std::vector<float>> train;
	std::vector<std::vector<std::vector<float>>> epocs;

	std::vector<float> ave_vec;
	std::vector<std::vector<float>> rotation;
	std::vector<float> sdev;

	train = LoadTrains("sample\\train\\convImg2Txt.txt",' ');
	ave_vec = LoadAverageVector("sample\\train\\average_vector.txt");
	rotation = LoadRotation("sample\\train\\rotation.txt");
	sdev = LoadStandardDev("sample\\train\\sdev.txt");


	map_width = MAP_WIDTH;
	map_height = MAP_HEIGHT;
	vec_dim = ave_vec.size();

	BLSOM blsom = BLSOM(vec_dim, map_width);
	blsom.Init(sdev[0], sdev[1], rotation[0].data(), rotation[1].data(), ave_vec.data());
	blsom.SetTrainingData(train);
	blsom.InitMapWeight(INIT_BATCH);

	/* Get initial map */
	som = blsom.GetSOMMap();
	WriteSOMMAP("sample\\result\\init_batch_map.txt", som, vec_dim, map_width, blsom.MapHeight());

	/* Get initial umatrix */
	umatrix = blsom.GetUMatrix();
	WriteUmatrix("sample\\result\\init_umatrix.txt", umatrix);


	/* Learning */
	blsom.Learning(50);

	/* Get Learned Map */
	som = blsom.GetSOMMap();
	WriteSOMMAP("sample\\result\\result_batch_map.txt", som, vec_dim, map_width, blsom.MapHeight());

	/* Get Umatrix */
	umatrix = blsom.GetUMatrix();
	WriteUmatrix("sample\\result\\result_umatrix.txt", umatrix);

	return 0;
}